#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "Circuit.h"

__device__
bool evaluateGate(bool *values, Gate gate) {
    // value "true" is 1, "false" is 0
    bool and_gate = true; // AND & NAND gate result
    bool or_gate = false; // OR & NOR gate result
    bool xor_gate = values[gate.inputs[0]]; // XOR & XNOR gate result
    for (size_t i = 0; i < gate.inputs.size(); i++) {
        if (!values[gate.inputs[i]]) {
            and_gate = false; // if any value is zero, result is zero
        } else {
            or_gate = true; // if any value is one, result is zero
        }
        xor_gate = xor_gate ^ values[gate.inputs[i]];
    }
    if (gate.type == "BUFF") {
        return values[gate.inputs[0]];
    } else if (gate.type == "NOT") {
        return !values[gate.inputs[0]];
    } else if (gate.type == "AND") {
        return and_gate;
    } else if (gate.type == "NAND") {
        return !and_gate;
    } else if (gate.type == "OR") {
        return or_gate;
    } else if (gate.type == "NOR") {
        return !or_gate;
    } else if (gate.type == "XOR") {
        return xor_gate;
    } else if (gate.type == "XNOR") {
        return !xor_gate;
    }
}

__global__ void
evaluateGates_kernel(Gate *gates, bool *testcase,
                     int depth, int *gatePara, int *gateParaSize, int *gateParaStartIdx,
                     int numOutput, int *outputId, bool *outputVal, bool *detected) {
    int numSignal = blockDim.x;
    int numTestcase = blockDim.y;
    int gateIdx = threadIdx.x; 
    int testcaseIdx = blockIdx.y;
    int faultIdx = blockIdx.x;

    extern __shared__ bool values[];
    for (int i = 0; i < depth; i++) {
        if (gateIdx < gateParaSize[i]) {
            int gateId = gatePara[gateParaStartIdx[i] + gateIdx];
            if (i == 0) {
                // Assign testcase to input values
                values[gateId] = testcase[testcaseIdx * gateParaSize[0] + gateIdx];
                if (gateId == faultIdx) {
                    values[gateId] = !values[gateId];
                }
            }
            // Evaluate gates
            else {
                bool gateValue = evaluateGate(values, gates[gateId]);
                values[gateId] = gateValue;
                if (gateId == faultIdx) {
                    values[gateId] = !gateValue;
                }
            }
            __syncthreads();
        }
    }
    // Save output values
    if (gateIdx == 0) {
        for (int i = 0; i < numOutput; i++) {
            detected[testcaseIdx * numSignal + faultIdx] = values[outputId[i]] != outputVal[testcaseIdx * numOutput + i];
        }
    }
}

bool *
ParaFaultSim(int numSignal, int numInput, Gate *gates, int numTestcase, bool *testcase, int depth, int maxGatePara, int *gatePara, int *gateParaSize, int *gateParaStartIdx, int numOutput, int *outputId, bool *outputVal) {

    Gate *device_gates; // 1D gates[signalID]
    bool *device_testcase; // 2D test[testID][inputID]
    int *device_gatePara; // 1D gatePara[signalID]
    int *device_gateParaSize; // 1D gateParaSize[depth]
    int *device_gateParaStartIdx; // 1D gateParaStartIdx[depth]
    int *device_outputId; // 1D signalID[outputID]
    bool *device_outputVal; // correct output values, 2D outputVal[testID][outputID]
    bool *device_detected; // 2D detected[testID][faultID]

    // Allocate device memory buffers on the GPU using hipMalloc
    hipMalloc(&device_gates, sizeof(Gate) * numSignal);
    hipMalloc(&device_testcase, sizeof(bool) * numTestcase * numInput);
    hipMalloc(&device_gatePara, sizeof(int) * numSignal);
    hipMalloc(&device_gateParaSize, sizeof(int) * depth);
    hipMalloc(&device_gateParaStartIdx, sizeof(int) * depth);
    hipMalloc(&device_outputId, sizeof(int) * numOutput);
    hipMalloc(&device_outputVal, sizeof(int) * numTestcase * numOutput);
    hipMalloc(&device_detected, sizeof(bool) * numTestcase * numSignal);

    // Start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    // Copy input arrays to the GPU using hipMemcpy
    hipMemcpy(device_gates, gates, sizeof(Gate) * numSignal, hipMemcpyHostToDevice);
    hipMemcpy(device_testcase, testcase, sizeof(bool) * numTestcase * numInput, hipMemcpyHostToDevice);
    hipMemcpy(device_gatePara, gatePara, sizeof(int) * numSignal, hipMemcpyHostToDevice);
    hipMemcpy(device_gateParaSize, gateParaSize, sizeof(int) * depth, hipMemcpyHostToDevice);
    hipMemcpy(device_gateParaStartIdx, gateParaStartIdx, sizeof(int) * depth, hipMemcpyHostToDevice);
    hipMemcpy(device_outputId, outputId, sizeof(int) * numOutput, hipMemcpyHostToDevice);
    hipMemcpy(device_outputVal, outputVal, sizeof(int) * numTestcase * numOutput, hipMemcpyHostToDevice);

    // Compute number of blocks and threads per block
    const int threadsPerBlock = maxGatePara;
    const int blocksX = numSignal;
    const int blocksY = numTestcase;
    
    dim3 gridDim(blocksX, blocksY);

    // Run kernel
    double startTimeKernel = CycleTimer::currentSeconds();
    evaluateGates_kernel<<<gridDim, threadsPerBlock, numSignal>>>
                    (device_gates, device_testcase, depth, device_gatePara, device_gateParaSize, device_gateParaStartIdx, numOutput, device_outputId, device_outputVal, device_detected);
    double endTimeKernel = CycleTimer::currentSeconds();

    bool *detected;
    // Copy result from GPU using hipMemcpy
    hipMemcpy(detected, device_detected, sizeof(int) * numTestcase * numSignal, hipMemcpyDeviceToHost);

    // End timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms", 1000.f * overallDuration);
    double overallDurationKernel = endTimeKernel - startTimeKernel;
    printf("Kernel: %.3f ms", 1000.f * overallDurationKernel);

    // Free memory buffers on the GPU
    hipFree(device_gates);
    hipFree(device_testcase);
    hipFree(device_gatePara);
    hipFree(device_gateParaSize);
    hipFree(device_gateParaStartIdx);
    hipFree(device_outputId);
    hipFree(device_outputVal);
    hipFree(device_detected);

    return detected;
}
#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "Circuit.h"

__device__
bool evaluateGate(bool *values, GATETYPE gateType, int *gateInput, int gateInputSize, int gateInputStartIdx) {
    // value "true" is 1, "false" is 0
    bool and_gate = true; // AND & NAND gate result
    bool or_gate = false; // OR & NOR gate result
    bool xor_gate = values[gateInput[gateInputStartIdx]]; // XOR & XNOR gate result
    for (size_t i = 0; i < gateInputSize; i++) {
        if (!values[gateInput[gateInputStartIdx + i]]) {
            and_gate = false; // if any value is zero, result is zero
        } else {
            or_gate = true; // if any value is one, result is one
        }
        if (i != 0) xor_gate = xor_gate ^ values[gateInput[gateInputStartIdx + i]];
    }
    if (gateType == BUFF) {
        return values[gateInput[gateInputStartIdx]];
    } else if (gateType == NOT) {
        return !values[gateInput[gateInputStartIdx]];
    } else if (gateType == AND) {
        return and_gate;
    } else if (gateType == NAND) {
        return !and_gate;
    } else if (gateType == OR) {
        return or_gate;
    } else if (gateType == NOR) {
        return !or_gate;
    } else if (gateType == XOR) {
        return xor_gate;
    } else if (gateType == XNOR) {
        return !xor_gate;
    }
}

__global__ void
evaluateGates_GatePara_kernel(GATETYPE *gateType, int *gateInput, int *gateInputSize, int *gateInputStartIdx,
                     bool *testcase, int depth, int *gatePara, int *gateParaSize, int *gateParaStartIdx,
                     int numOutput, int *outputId, bool *outputVal, bool *detected) {
    int numSignal = gridDim.x;
    // int numTestcase = gridDim.y;
    int gateIdx = threadIdx.x; 
    int testcaseIdx = blockIdx.y;
    int faultIdx = blockIdx.x;

    extern __shared__ bool values[]; // Values shared among threads, per fault per testcase
    for (int i = 0; i < depth; i++) {
        // Only gateParaSize[i] number of threads are processed in parallel at once
        if (gateIdx < gateParaSize[i]) {
            int gateId = gatePara[gateParaStartIdx[i] + gateIdx]; // signalID
            // Assign testcase to input values
            if (i == 0) {
                values[gateId] = testcase[testcaseIdx * gateParaSize[0] + gateIdx];
                // Input faults
                if (gateId == faultIdx) {
                    values[gateId] = !values[gateId];
                }
            }
            // Evaluate gates
            else {
                values[gateId] = evaluateGate(values, gateType[gateId], gateInput, gateInputSize[gateId], gateInputStartIdx[gateId]);
                // Signal faults
                if (gateId == faultIdx) {
                    values[gateId] = !values[gateId];
                }
            }
        }
        __syncthreads(); // Sync all threads between gatePara
    }
    
    // Save output values
    if (gateIdx == 0) {
        for (int i = 0; i < numOutput; i++) {
            // Fault can be detected if any faulty circuit output values are different from good circuit output values
            detected[testcaseIdx * numSignal + faultIdx] |= (values[outputId[i]] != outputVal[testcaseIdx * numOutput + i]);
        }
    }
}

bool *
ParaFaultSim_GatePara(int numSignal, int numInput, GATETYPE *gateType, int numGateInput, int *gateInput, int *gateInputSize, int *gateInputStartIdx, int numTestcase, bool *testcase, int depth, int maxGatePara, int *gatePara, int *gateParaSize, int *gateParaStartIdx, int numOutput, int *outputId, bool *outputVal) {

    // Start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    GATETYPE *device_gateType; // 1D gateType[signalID]
    int *device_gateInput; // 2D gateInput[gateID][inputID]
    int *device_gateInputSize; // 1D gateInputSize[signalID]
    int *device_gateInputStartIdx; // 1D gateInputStartIdx[signalID]
    bool *device_testcase; // 2D test[testID][inputID]
    int *device_gatePara; // 1D gatePara[signalID]
    int *device_gateParaSize; // 1D gateParaSize[depth]
    int *device_gateParaStartIdx; // 1D gateParaStartIdx[depth]
    int *device_outputId; // 1D signalID[outputID]
    bool *device_outputVal; // correct output values, 2D outputVal[testID][outputID]
    bool *device_detected; // 2D detected[testID][faultID]

    // Allocate device memory buffers on the GPU using hipMalloc
    hipMalloc(&device_gateType, sizeof(GATETYPE) * numSignal);
    hipMalloc(&device_gateInput, sizeof(int) * numGateInput);
    hipMalloc(&device_gateInputSize, sizeof(int) * numSignal);
    hipMalloc(&device_gateInputStartIdx, sizeof(int) * numSignal);
    hipMalloc(&device_testcase, sizeof(bool) * numTestcase * numInput);
    hipMalloc(&device_gatePara, sizeof(int) * numSignal);
    hipMalloc(&device_gateParaSize, sizeof(int) * depth);
    hipMalloc(&device_gateParaStartIdx, sizeof(int) * depth);
    hipMalloc(&device_outputId, sizeof(int) * numOutput);
    hipMalloc(&device_outputVal, sizeof(int) * numTestcase * numOutput);
    hipMalloc(&device_detected, sizeof(bool) * numTestcase * numSignal);

    // Copy input arrays to the GPU using hipMemcpy
    hipMemcpy(device_gateType, gateType, sizeof(GATETYPE) * numSignal, hipMemcpyHostToDevice);
    hipMemcpy(device_gateInput, gateInput, sizeof(int) * numGateInput, hipMemcpyHostToDevice);
    hipMemcpy(device_gateInputSize, gateInputSize, sizeof(int) * numSignal, hipMemcpyHostToDevice);
    hipMemcpy(device_gateInputStartIdx, gateInputStartIdx, sizeof(int) * numSignal, hipMemcpyHostToDevice);
    hipMemcpy(device_testcase, testcase, sizeof(bool) * numTestcase * numInput, hipMemcpyHostToDevice);
    hipMemcpy(device_gatePara, gatePara, sizeof(int) * numSignal, hipMemcpyHostToDevice);
    hipMemcpy(device_gateParaSize, gateParaSize, sizeof(int) * depth, hipMemcpyHostToDevice);
    hipMemcpy(device_gateParaStartIdx, gateParaStartIdx, sizeof(int) * depth, hipMemcpyHostToDevice);
    hipMemcpy(device_outputId, outputId, sizeof(int) * numOutput, hipMemcpyHostToDevice);
    hipMemcpy(device_outputVal, outputVal, sizeof(int) * numTestcase * numOutput, hipMemcpyHostToDevice);

    // Compute number of blocks and threads per block
    const int threadsPerBlock = maxGatePara;
    const int blocksX = numSignal;
    const int blocksY = numTestcase;
    
    dim3 gridDim(blocksX, blocksY);

    // End timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    // Run kernel
    double startTimeKernel = CycleTimer::currentSeconds();
    evaluateGates_GatePara_kernel<<<gridDim, threadsPerBlock, sizeof(bool)*numSignal>>>
                    (device_gateType, device_gateInput, device_gateInputSize, device_gateInputStartIdx,
                     device_testcase, depth, device_gatePara, device_gateParaSize, device_gateParaStartIdx, numOutput, device_outputId, device_outputVal, device_detected);
    double endTimeKernel = CycleTimer::currentSeconds();

    bool *detected = new bool[numTestcase * numSignal];
    // Copy result from GPU using hipMemcpy
    hipMemcpy(detected, device_detected, sizeof(bool) * numTestcase * numSignal, hipMemcpyDeviceToHost);

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("CUDA mem alloc & copy: %.3f ms\n", 1000.f * overallDuration);
    double overallDurationKernel = endTimeKernel - startTimeKernel;
    printf("Kernel: %.3f ms\n", 1000.f * overallDurationKernel);

    // Free memory buffers on the GPU
    hipFree(device_gateType);
    hipFree(device_gateInput);
    hipFree(device_gateInputSize);
    hipFree(device_gateInputStartIdx);
    hipFree(device_testcase);
    hipFree(device_gatePara);
    hipFree(device_gateParaSize);
    hipFree(device_gateParaStartIdx);
    hipFree(device_outputId);
    hipFree(device_outputVal);
    hipFree(device_detected);

    return detected;
}

__global__ void
evaluateGates_TestcasePara_kernel(int numSignal, GATETYPE *gateType, int *gateInput, int *gateInputSize, int *gateInputStartIdx,
                     bool *testcase, int *gatePara, int numInput, int numOutput, int *outputId, bool *outputVal, bool *detected) {
    int partitionIdx = blockIdx.x;
    int faultId = blockDim.x * partitionIdx + threadIdx.x; // Global fault id within a testcase
    int faultIdx = threadIdx.x; // Local partition fault index
    int testcaseIdx = blockIdx.y;

    extern __shared__ bool values[]; // numFault * numSignal
    if (faultId < numSignal) {
        for (int i = 0; i < numSignal; i++) {
            int gateId = gatePara[i]; // signalID
            int valueIdx = faultIdx * numSignal + gateId; 
            // Assign testcase to input values
            if (gateId < numInput) {
                values[valueIdx] = testcase[testcaseIdx * numInput + gateId];
                // Input faults
                if (gateId == faultId) {
                    values[valueIdx] = !values[valueIdx];
                }
            }
            // Evaluate gates
            else {
                values[valueIdx] = evaluateGate((values + faultIdx * numSignal), gateType[gateId], gateInput, gateInputSize[gateId], gateInputStartIdx[gateId]);
                // Signal faults
                if (gateId == faultId) {
                    values[valueIdx] = !values[valueIdx];
                }
            }
        }
    
        // Save output values
        for (int i = 0; i < numOutput; i++) {
            // Fault can be detected if any faulty circuit output values are different from good circuit output values
            detected[testcaseIdx * numSignal + faultId] |= (values[faultIdx * numSignal + outputId[i]] != outputVal[testcaseIdx * numOutput + i]);
        }
    }
}

bool *
ParaFaultSim_TestcasePara(int numSignal, int numInput, GATETYPE *gateType, int numGateInput, int *gateInput, int *gateInputSize, int *gateInputStartIdx, int numTestcase, bool *testcase, int *gatePara, int numOutput, int *outputId, bool *outputVal) {

    // Start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    GATETYPE *device_gateType; // 1D gateType[signalID]
    int *device_gateInput; // 2D gateInput[gateID][inputID]
    int *device_gateInputSize; // 1D gateInputSize[signalID]
    int *device_gateInputStartIdx; // 1D gateInputStartIdx[signalID]
    bool *device_testcase; // 2D test[testID][inputID]
    int *device_gatePara; // 1D gatePara[signalID]
    int *device_outputId; // 1D signalID[outputID]
    bool *device_outputVal; // correct output values, 2D outputVal[testID][outputID]
    bool *device_detected; // 2D detected[testID][faultID]

    // Allocate device memory buffers on the GPU using hipMalloc
    hipMalloc(&device_gateType, sizeof(GATETYPE) * numSignal);
    hipMalloc(&device_gateInput, sizeof(int) * numGateInput);
    hipMalloc(&device_gateInputSize, sizeof(int) * numSignal);
    hipMalloc(&device_gateInputStartIdx, sizeof(int) * numSignal);
    hipMalloc(&device_testcase, sizeof(bool) * numTestcase * numInput);
    hipMalloc(&device_gatePara, sizeof(int) * numSignal);
    hipMalloc(&device_outputId, sizeof(int) * numOutput);
    hipMalloc(&device_outputVal, sizeof(int) * numTestcase * numOutput);
    hipMalloc(&device_detected, sizeof(bool) * numTestcase * numSignal);

    // Copy input arrays to the GPU using hipMemcpy
    hipMemcpy(device_gateType, gateType, sizeof(GATETYPE) * numSignal, hipMemcpyHostToDevice);
    hipMemcpy(device_gateInput, gateInput, sizeof(int) * numGateInput, hipMemcpyHostToDevice);
    hipMemcpy(device_gateInputSize, gateInputSize, sizeof(int) * numSignal, hipMemcpyHostToDevice);
    hipMemcpy(device_gateInputStartIdx, gateInputStartIdx, sizeof(int) * numSignal, hipMemcpyHostToDevice);
    hipMemcpy(device_testcase, testcase, sizeof(bool) * numTestcase * numInput, hipMemcpyHostToDevice);
    hipMemcpy(device_gatePara, gatePara, sizeof(int) * numSignal, hipMemcpyHostToDevice);
    hipMemcpy(device_outputId, outputId, sizeof(int) * numOutput, hipMemcpyHostToDevice);
    hipMemcpy(device_outputVal, outputVal, sizeof(int) * numTestcase * numOutput, hipMemcpyHostToDevice);

    // Compute number of blocks and threads per block
    const int partition = 256;
    const int threadsPerBlock = (numSignal + partition - 1) / partition;
    dim3 gridDim (partition, numTestcase);

    // End timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    // Run kernel
    double startTimeKernel = CycleTimer::currentSeconds();
    evaluateGates_TestcasePara_kernel<<<gridDim, threadsPerBlock, sizeof(bool)*((numSignal+partition-1)/partition) * numSignal>>>
                    (numSignal, device_gateType, device_gateInput, device_gateInputSize, device_gateInputStartIdx,
                     device_testcase, device_gatePara, numInput, numOutput, device_outputId, device_outputVal, device_detected);
    double endTimeKernel = CycleTimer::currentSeconds();

    bool *detected = new bool[numTestcase * numSignal];
    // Copy result from GPU using hipMemcpy
    hipMemcpy(detected, device_detected, sizeof(bool) * numTestcase * numSignal, hipMemcpyDeviceToHost);

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("CUDA mem alloc & copy: %.3f ms\n", 1000.f * overallDuration);
    double overallDurationKernel = endTimeKernel - startTimeKernel;
    printf("Kernel: %.3f ms\n", 1000.f * overallDurationKernel);

    // Free memory buffers on the GPU
    hipFree(device_gateType);
    hipFree(device_gateInput);
    hipFree(device_gateInputSize);
    hipFree(device_gateInputStartIdx);
    hipFree(device_testcase);
    hipFree(device_gatePara);
    hipFree(device_outputId);
    hipFree(device_outputVal);
    hipFree(device_detected);

    return detected;
}